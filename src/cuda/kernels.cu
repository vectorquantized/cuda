#include "hip/hip_runtime.h"
#include "kernels.h"

__global__ void saxpy_grid_strided(float a, float* b, float* c, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i=index; i< N; i += stride) {
        c[i] += a * b[i];
    }
}

__global__ void saxpy(float a, float* b, float* c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        c[i] += a * b[i];
    }
}

__global__ void conv1d(float* matrix, float* conv_mask, float* output, int mask_width, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float p_value = 0.0f;
    int start = idx - mask_width / 2;
    for (int j = 0; j < mask_width; ++j) {
        if (start + j >= 0 && start + j < width) {
            p_value += matrix[start + j] * conv_mask[j];
        }
    }
    output[idx] = p_value;
}

__global__ void rgbToGrayScale(float *in, float *out, int width, int height) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < width && row < height) {
        int gray_offset = row * width + col;
        int rgb_offset = gray_offset * CHANNELS;
        float r = in[rgb_offset];
        float g = in[rgb_offset + 1];
        float b = in[rgb_offset + 2];
        out[gray_offset] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
}
