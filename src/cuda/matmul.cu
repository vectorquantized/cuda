#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <hipblas.h>
#include "kernels.h"
#include "csrc/matrix.h"
#include "csrc/init_utils.h"
#include "cpu/cpu_kernels.h"
#include "csrc/utils.h"

int main() {
    
    int M = 1024;
    int K = 1024;
    int N = 1024;
    Matrix mat1_h(M, K, random_init);
    Matrix mat2_h(K, N, random_init);
    float* out_h = new float[M * N];
    float* out_cpu = new float[M * N];
    matmul_cpu(mat1_h.data.get(), mat2_h.data.get(), out_cpu, M, K, N);
    float *mat1_d, *mat2_d, *out_d;

    CUDA_ERROR_CHECK(hipMalloc((void**) &mat1_d, M * K * sizeof(float)));
    hipMalloc((void**) &mat2_d, K * N * sizeof(float));
    hipMalloc((void**) &out_d, M * N * sizeof(float));

    hipMemcpy(mat1_d, mat1_h.data.get(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mat2_d, mat2_h.data.get(), K * N * sizeof(float), hipMemcpyHostToDevice);

    int block_size_x = 16;
    int block_size_y = 16;

    dim3 threads_per_block(block_size_x, 
                           block_size_y);

    dim3 blocks_per_grid((N + block_size_x - 1) / block_size_x, 
                         (M + block_size_y - 1) / block_size_y);


    float alpha = 1.0f;
    float beta = 0.0f;
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record the start event
    hipEventRecord(start);
    matmul_cuda<<<blocks_per_grid, threads_per_block>>>(mat1_d, mat2_d, out_d, M, K, N);
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, mat2_d, N, mat1_d, K, &beta, out_d, N);
    // hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K, &alpha, mat2_d, K, mat1_d, M, &beta, out_d, N);
    
    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // hipDeviceSynchronize();
    hipMemcpy(out_h, out_d, M * N * sizeof(float), hipMemcpyDeviceToHost);

     // Calculate the elapsed time between the start and stop events
     float milliseconds = 0;
     hipEventElapsedTime(&milliseconds, start, stop);
 
     std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    if (matrix::compare_matrices(out_h, out_cpu, M, N)) {
        std::cout << "The CUDA kernel's result matches the CPU result." << std::endl;
    } else {
        std::cerr << "The CUDA kernel's result does NOT match the CPU result." << std::endl;
    }

    delete [] out_h;
    delete [] out_cpu;
    hipFree(mat1_d);
    hipFree(mat2_d);
    hipFree(out_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
}