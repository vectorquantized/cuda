#include <iostream>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <hipblas.h>
#include "kernels.h"
#include "csrc/matrix.h"
#include "csrc/init_utils.h"
#include "cpu/cpu_kernels.h"
#include "csrc/utils.h"


void conv1d_kernel(float* matrix_d, float* conv_mask_d, float* output_d, int mask_width, int width) {
    TIMED_CUDA_FUNCTION();
    int block_size = 256;
    dim3 threads_per_block(block_size, 1, 1);
    dim3 blocks_per_grid((width + (block_size - 1))/block_size, 1, 1);

    conv1d<<<blocks_per_grid, threads_per_block>>>(matrix_d, conv_mask_d, output_d, mask_width, width);
    hipDeviceSynchronize();

}

void conv1d_kernel_invocation() {
    int N = 1 << 20;
    std::vector<float> matrix_h(N);
    random_init(matrix_h.data(), N);
    std::vector<float> conv_mask_h = {0.2, 0.5, 0.2};
    int mask_width = conv_mask_h.size();
    int width = matrix_h.size();
    std::vector<float> cpu_output(width);

    conv1d_cpu(matrix_h, conv_mask_h, cpu_output, mask_width, width);

    float *matrix_d, *conv_mask_d, *output_d;

    float* output_h = new float[width];

    CUDA_ERROR_CHECK(hipMalloc((void**) &matrix_d, sizeof(float) * width));
    CUDA_ERROR_CHECK(hipMalloc((void**) &conv_mask_d, sizeof(float) * mask_width));
    CUDA_ERROR_CHECK(hipMalloc((void**) &output_d, sizeof(float) * width));

    CUDA_ERROR_CHECK(hipMemcpy(matrix_d, matrix_h.data(), sizeof(float) * width, hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(hipMemcpy(conv_mask_d, conv_mask_h.data(), sizeof(float) * mask_width, hipMemcpyHostToDevice));

    conv1d_kernel(matrix_d, conv_mask_d, output_d, mask_width, width);
    
    CUDA_ERROR_CHECK(hipMemcpy(output_h, output_d, sizeof(float) * width, hipMemcpyDeviceToHost));

    if (utils::compare_vectors(output_h, cpu_output.data(), N)) {
        std::cout << "The CUDA kernel's result matches the CPU result." << std::endl;
    } else {
        std::cerr << "The CUDA kernel's result does NOT match the CPU result." << std::endl;
    }
    
    delete[] output_h;
    hipFree(matrix_d);
    hipFree(conv_mask_d);
    hipFree(output_d);
}

int main() {
    
    conv1d_kernel_invocation();
    return 0;
}

int main2() {
    
    int M = 1024;
    int K = 1024;
    int N = 1024;
    Matrix mat1_h(M, K, random_init);
    Matrix mat2_h(K, N, random_init);
    float* out_h = new float[M * N];
    float* out_cpu = new float[M * N];
    matmul_cpu(mat1_h.data.get(), mat2_h.data.get(), out_cpu, M, K, N);
    float *mat1_d, *mat2_d, *out_d;

    CUDA_ERROR_CHECK(hipMalloc((void**) &mat1_d, M * K * sizeof(float)));
    hipMalloc((void**) &mat2_d, K * N * sizeof(float));
    hipMalloc((void**) &out_d, M * N * sizeof(float));

    hipMemcpy(mat1_d, mat1_h.data.get(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mat2_d, mat2_h.data.get(), K * N * sizeof(float), hipMemcpyHostToDevice);

    int block_size_x = 16;
    int block_size_y = 16;

    dim3 threads_per_block(block_size_x, 
                           block_size_y);

    dim3 blocks_per_grid((N + block_size_x - 1) / block_size_x, 
                         (M + block_size_y - 1) / block_size_y);


    float alpha = 1.0f;
    float beta = 0.0f;
    // hipblasHandle_t handle;
    // hipblasCreate(&handle);
    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Record the start event
    hipEventRecord(start);
    matmul_cuda<<<blocks_per_grid, threads_per_block>>>(mat1_d, mat2_d, out_d, M, K, N);
    // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, mat2_d, N, mat1_d, K, &beta, out_d, N);
    // hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K, &alpha, mat2_d, K, mat1_d, M, &beta, out_d, N);
    
    // Record the stop event
    hipEventRecord(stop);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // hipDeviceSynchronize();
    hipMemcpy(out_h, out_d, M * N * sizeof(float), hipMemcpyDeviceToHost);

     // Calculate the elapsed time between the start and stop events
     float milliseconds = 0;
     hipEventElapsedTime(&milliseconds, start, stop);
 
     std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    if (matrix::compare_matrices(out_h, out_cpu, M, N)) {
        std::cout << "The CUDA kernel's result matches the CPU result." << std::endl;
    } else {
        std::cerr << "The CUDA kernel's result does NOT match the CPU result." << std::endl;
    }

    delete [] out_h;
    delete [] out_cpu;
    hipFree(mat1_d);
    hipFree(mat2_d);
    hipFree(out_d);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
}