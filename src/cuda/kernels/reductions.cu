#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "reductions.h"
#include "csrc/init_utils.h"
#include "cuda/utils/cuda_utils.h"
#include <numeric>
#include <iostream>


__global__ void add(float* input, float* output, int size) {

    extern __shared__ float s_data[];
    unsigned int t_idx = threadIdx.x;
    unsigned int g_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (g_idx < size) {
        s_data[t_idx] = input[g_idx];
    } else {
        s_data[t_idx] = 0.0f;
    }
    __syncthreads(); // barrier sync, we wait for all the data to be loaded in the SMEM.

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (t_idx < stride) {
            s_data[t_idx] += s_data[t_idx + stride];
        }
        __syncthreads();
    }
    if (t_idx == 0) {
        output[blockIdx.x] = s_data[t_idx];
    }
}

bool close(float a, float b, float rtol = 1e-5, float atol = 1e-8) {
    return std::fabs(a - b) <= (atol + rtol * std::fabs(b));
}


namespace reduction_kernels {

void add_kernel_launch(float* input, float* output, int size, int threads_per_block, int blocks_per_grid) {
    TIMED_CUDA_FUNCTION();
    int shared_memory_size = threads_per_block * sizeof(float);
    add<<<blocks_per_grid, threads_per_block, shared_memory_size>>>(input, output, size);
    hipDeviceSynchronize();
}

void launch(std::string name) {
    int M = 8192;
    int threads_per_block = 256;
    int blocks_per_grid = (M + (threads_per_block - 1)) / threads_per_block;
    float* output_h = new float[blocks_per_grid]();
    std::vector<float> input_vec(M);

    random_init(input_vec.data(), M);

    float *input_d, *output_d;

    CUDA_ERROR_CHECK(hipMalloc((void**) &input_d, sizeof(float) * M));
    CUDA_ERROR_CHECK(hipMalloc((void**) &output_d, sizeof(float) * blocks_per_grid));

    CUDA_ERROR_CHECK(hipMemcpy(input_d, input_vec.data(), sizeof(float) * M, hipMemcpyHostToDevice));

    add_kernel_launch(input_d, output_d, M, threads_per_block, blocks_per_grid);
    
    CUDA_ERROR_CHECK(hipMemcpy(output_h, output_d, sizeof(float) * blocks_per_grid, hipMemcpyDeviceToHost));

    std::cout.precision(10);
    float cpu_sum = std::accumulate(input_vec.begin(), input_vec.end(), 0.0f);
    std::cout << "cpu: " << cpu_sum << std::endl;
    float gpu_sum = std::accumulate(output_h, output_h + blocks_per_grid, 0.0f);
    std::cout << "gpu: " << gpu_sum << std::endl;
    std::cout << "diff: " << fabs(cpu_sum - gpu_sum) << std::endl;
    if (close(cpu_sum, gpu_sum)) {
        std::cout << "CUDA kernel's result matches the CPU result." << std::endl;
    } else {
        std::cout << "CUDA kernel's result does NOT match the CPU result." << std::endl;
    }

    CUDA_ERROR_CHECK(hipFree(input_d));
    CUDA_ERROR_CHECK(hipFree(output_d));
    delete output_h;


}
}
