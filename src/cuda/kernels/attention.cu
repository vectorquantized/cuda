
#include <iostream>
#include <vector>
#include <iomanip>
#include "csrc/utils.h"
#include "csrc/matrix.h"
#include "csrc/init_utils.h"
#include "cpu/cpu_kernels.h"
#include "cuda/utils/cuda_utils.h"
#include "attention.h"



namespace attn {
void launch() {

    int B = 16; //batch size
    int S = 16; // sequence length
    int D_model = 8; // dim of Q, K and V
    
    //initialize host q, k, v
    // use 3d tensor implementation. 
    CUDA_ERROR_CHECK(hipMalloc((void**) &q_d, B * S * D_model * sizeof(float)));
    CUDA_ERROR_CHECK(hipMalloc((void**) &k_d, B * S * D_model * sizeof(float)));
    CUDA_ERROR_CHECK(hipMalloc((void**) &v_d, B * S * D_model * sizeof(float)));

    hipFree(q_d);
    hipFree(k_d);
    hipFree(v_d);
}
}
